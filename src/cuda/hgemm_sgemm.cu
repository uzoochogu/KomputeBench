#include "hip/hip_runtime.h"
// sample cuBLAS program derived from
// https://github.com/Infatoshi/cuda-course/
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "../cuda_utilities.hpp"
#include "../data.hpp"  // mat A, B
#include "../utils.hpp"

#undef PRINT_MATRIX
#define PRINT_MATRIX(mat, rows, cols)                                   \
  for (int i = 0; i < rows; i++) {                                      \
    for (int j = 0; j < cols; j++) printf("%8.3f ", mat[i * cols + j]); \
    printf("\n");                                                       \
  }                                                                     \
  printf("\n");

int main() {
  float C_cpu[M * N], C_cublas_s[M * N], C_cublas_h[M * N];

  // CPU matmul using ijk loop order
  cpu_matmul_ikj(A, B, std::span<float>(C_cpu, M * N), M, K, N);

  // CUDA setup
  hipblasHandle_t handle;
  CHECK_CUBLAS(hipblasCreate(&handle));

  float *d_A, *d_B, *d_C;
  CHECK_CUDA(hipMalloc(&d_A, M * K * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_B, K * N * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_C, M * N * sizeof(float)));

  CHECK_CUDA(
      hipMemcpy(d_A, A.data(), M * K * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(
      hipMemcpy(d_B, B.data(), K * N * sizeof(float), hipMemcpyHostToDevice));

  // cuBLAS SGEMM
  float alpha = 1.0f, beta = 0.0f;
  CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha,
                           d_B, N, d_A, K, &beta, d_C, N));
  CHECK_CUDA(hipMemcpy(C_cublas_s, d_C, M * N * sizeof(float),
                        hipMemcpyDeviceToHost));

  // cuBLAS HGEMM
  half *d_A_h, *d_B_h, *d_C_h;
  CHECK_CUDA(hipMalloc(&d_A_h, M * K * sizeof(half)));
  CHECK_CUDA(hipMalloc(&d_B_h, K * N * sizeof(half)));
  CHECK_CUDA(hipMalloc(&d_C_h, M * N * sizeof(half)));

  // Convert to half precision on CPU
  half A_h[M * K], B_h[K * N];
  for (int i = 0; i < M * K; i++) {
    A_h[i] = __float2half(A[i]);
  }
  for (int i = 0; i < K * N; i++) {
    B_h[i] = __float2half(B[i]);
  }

  // Copy half precision data to device
  CHECK_CUDA(
      hipMemcpy(d_A_h, A_h, M * K * sizeof(half), hipMemcpyHostToDevice));
  CHECK_CUDA(
      hipMemcpy(d_B_h, B_h, K * N * sizeof(half), hipMemcpyHostToDevice));

  __half alpha_h = __float2half(1.0f), beta_h = __float2half(0.0f);
  CHECK_CUBLAS(hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha_h,
                           d_B_h, N, d_A_h, K, &beta_h, d_C_h, N));

  // Copy result back to host and convert to float
  half C_h[M * N];
  CHECK_CUDA(
      hipMemcpy(C_h, d_C_h, M * N * sizeof(half), hipMemcpyDeviceToHost));
  for (int i = 0; i < M * N; i++) {
    C_cublas_h[i] = __half2float(C_h[i]);
  }

  // Print results
  printf("Matrix A (%dx%d):\n", M, K);
  PRINT_MATRIX(A.data(), M, K);
  printf("Matrix B (%dx%d):\n", K, N);
  PRINT_MATRIX(B.data(), K, N);
  printf("CPU Result (%dx%d):\n", M, N);
  PRINT_MATRIX(C_cpu, M, N);
  printf("cuBLAS SGEMM Result (%dx%d):\n", M, N);
  PRINT_MATRIX(C_cublas_s, M, N);
  printf("cuBLAS HGEMM Result (%dx%d):\n", M, N);
  PRINT_MATRIX(C_cublas_h, M, N);

  // Compare results
  if (compareResults(std::span<float>(C_cpu, M * N),
                     std::span<float>(C_cublas_s, M * N), 1e-3f)) {
    printf("cuBLAS SGEMM Result matches CPU Result\n");
  } else {
    printf("cuBLAS SGEMM Result does not match CPU Result\n");
  }

  // Clean up
  CHECK_CUDA(hipFree(d_A));
  CHECK_CUDA(hipFree(d_B));
  CHECK_CUDA(hipFree(d_C));
  CHECK_CUDA(hipFree(d_A_h));
  CHECK_CUDA(hipFree(d_B_h));
  CHECK_CUDA(hipFree(d_C_h));
  CHECK_CUBLAS(hipblasDestroy(handle));

  return 0;
}
